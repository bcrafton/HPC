
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include "string.h"
#include <sys/time.h>
#include <time.h>

#define DEFAULT_THRESHOLD  4000
#define DEFAULT_FILENAME "ansel3.ppm"

unsigned int* read_ppm(char* filename, int* image_width, int* image_height, int* maxval);
void write_ppm(char* filename, int image_width, int image_height, int maxval, int* pic);
__global__ void sobel(int* input_image, int* output_image, int image_width, int image_height, int thresh);

int main( int argc, char **argv )
{
	int thresh = DEFAULT_THRESHOLD;
	char *filename;
	filename = strdup( DEFAULT_FILENAME);

	if (argc > 1) 
	{
		if (argc == 3)  
		{ // filename AND threshold
			filename = strdup( argv[1]);
			thresh = atoi( argv[2] );
		}
		if (argc == 2) 
		{ // default file but specified threshhold
			thresh = atoi( argv[1] );
		}
		fprintf(stderr, "file %s    threshold %d\n", filename, thresh);
	}


	int image_width, image_height, maxval;
	unsigned int *host_input_image = read_ppm( filename, &image_width, &image_height, &maxval );

	int numbytes =  image_width * image_height * sizeof(int);
	int *host_result = (int *) malloc(numbytes);
	if (!host_result) 
	{
		fprintf(stderr, "sobel() unable to malloc %d bytes\n", numbytes);
		exit(-1); // fail
	}
	
	int* device_input_image;
	int* device_result;
	
	hipMalloc(&device_input_image, numbytes);
	hipMalloc(&device_result, numbytes);	
	
	hipMemcpy(device_input_image, host_input_image, numbytes, hipMemcpyHostToDevice);
	
	dim3 threads_per_block(32, 32);
	dim3 blocks_per_grid(ceil(image_width/32.0), ceil(image_height/32.0));
	
	float elapsed_time;
	hipEvent_t start_event, stop_event;
	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);
	hipEventRecord(start_event, 0);

	sobel<<<blocks_per_grid, threads_per_block>>>(device_input_image, device_result, image_width, image_height, thresh);
	
	hipEventRecord(stop_event, 0);
	hipEventSynchronize(stop_event);
	hipEventElapsedTime(&elapsed_time, start_event, stop_event);

	hipMemcpy(host_result, device_result, numbytes, hipMemcpyDeviceToHost);
	
	hipFree(device_input_image);
	hipFree(device_result);
	
	write_ppm( "result.ppm", image_width, image_height, 255, host_result);

	printf("Elapsed time: %f milliseconds\n", elapsed_time);
	
	return 0;
}

unsigned int *read_ppm( char *filename, int * image_width, int * image_height, int *maxval ){

  if ( !filename || filename[0] == '\0') {
    fprintf(stderr, "read_ppm but no file name\n");
    return NULL;  // fail
  }

  FILE *fp;

  fprintf(stderr, "read_ppm( %s )\n", filename);
  fp = fopen( filename, "rb");
  if (!fp)
    {
      fprintf(stderr, "read_ppm()    ERROR  file '%s' cannot be opened for reading\n", filename);
      return NULL; // fail

    }

  char chars[1024];
  //int num = read(fd, chars, 1000);
  int num = fread(chars, sizeof(char), 1000, fp);

  if (chars[0] != 'P' || chars[1] != '6')
    {
      fprintf(stderr, "Texture::Texture()    ERROR  file '%s' does not start with \"P6\"  I am expecting a binary PPM file\n", filename);
      return NULL;
    }

  unsigned int width, height, maxvalue;


  char *ptr = chars+3; // P 6 newline
  if (*ptr == '#') // comment line!
    {
      ptr = 1 + strstr(ptr, "\n");
    }

  num = sscanf(ptr, "%d\n%d\n%d",  &width, &height, &maxvalue);
  fprintf(stderr, "read %d things   width %d  height %d  maxval %d\n", num, width, height, maxvalue);
  *image_width = width;
  *image_height = height;
  *maxval = maxvalue;

  unsigned int *pic = (unsigned int *)malloc( width * height * sizeof(unsigned int));
  if (!pic) {
    fprintf(stderr, "read_ppm()  unable to allocate %d x %d unsigned ints for the picture\n", width, height);
    return NULL; // fail but return
  }

  // allocate buffer to read the rest of the file into
  int bufsize =  3 * width * height * sizeof(unsigned char);
  if ((*maxval) > 255) bufsize *= 2;
  unsigned char *buf = (unsigned char *)malloc( bufsize );
  if (!buf) {
    fprintf(stderr, "read_ppm()  unable to allocate %d bytes of read buffer\n", bufsize);
    return NULL; // fail but return
  }
  
  // TODO really read
  char duh[80];
  char *line = chars;

  // find the start of the pixel data.   no doubt stupid
  sprintf(duh, "%d\0", *image_width);
  line = strstr(line, duh);
  //fprintf(stderr, "%s found at offset %d\n", duh, line-chars);
  line += strlen(duh) + 1;

  sprintf(duh, "%d\0", *image_height);
  line = strstr(line, duh);
  //fprintf(stderr, "%s found at offset %d\n", duh, line-chars);
  line += strlen(duh) + 1;

  sprintf(duh, "%d\0", *maxval);
  line = strstr(line, duh);


  fprintf(stderr, "%s found at offset %d\n", duh, line - chars);
  line += strlen(duh) + 1;

  long offset = line - chars;
  //lseek(fd, offset, SEEK_SET); // move to the correct offset
  fseek(fp, offset, SEEK_SET); // move to the correct offset
  //long numread = read(fd, buf, bufsize);
  long numread = fread(buf, sizeof(char), bufsize, fp);
  fprintf(stderr, "Texture %s   read %ld of %ld bytes\n", filename, numread, bufsize);

  fclose(fp);


  int pixels = (*image_width) * (*image_height);
  int i;
  for (i=0; i<pixels; i++) pic[i] = (int) buf[3*i];  // red channel

  return pic; // success
}

void write_ppm( char *filename, int image_width, int image_height, int maxval, int *pic)
{
  FILE *fp;

  fp = fopen(filename, "w");
  if (!fp)
    {
      fprintf(stderr, "FAILED TO OPEN FILE '%s' for writing\n");
      exit(-1);
    }

  fprintf(fp, "P6\n");
  fprintf(fp,"%d %d\n%d\n", image_width, image_height, maxval);

  int numpix = image_width * image_height;
  int i;
  for (i=0; i<numpix; i++) {
    unsigned char uc = (unsigned char) pic[i];
    fprintf(fp, "%c%c%c", uc, uc, uc);
  }
  fclose(fp);

}

__global__ void sobel(int* input_image, int* output_image, int image_width, int image_height, int thresh)
{	
	int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int i = blockIdx.y * blockDim.y + threadIdx.y + 1;

	if( (i < (image_height-1)) && (j < (image_width-1)) )
	{
		int sum1, sum2, magnitude;
	
		int offset = i*image_width + j;

		sum1 =  input_image[ image_width * (i-1) + j+1 ] -     input_image[ image_width*(i-1) + j-1 ]
		+ 2 * input_image[ image_width * (i)   + j+1 ] - 2 * input_image[ image_width*(i)   + j-1 ]
		+     input_image[ image_width * (i+1) + j+1 ] -     input_image[ image_width*(i+1) + j-1 ];

		sum2 = input_image[ image_width * (i-1) + j-1 ] + 2 * input_image[ image_width * (i-1) + j ]  + input_image[ image_width * (i-1) + j+1 ]
		- input_image[image_width * (i+1) + j-1 ] - 2 * input_image[ image_width * (i+1) + j ] - input_image[ image_width * (i+1) + j+1 ];

		magnitude =  sum1*sum1 + sum2*sum2;

		if (magnitude > thresh)
			output_image[offset] = 255;
		else
			output_image[offset] = 0;
	}
}
