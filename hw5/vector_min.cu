
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
 
__global__ void vector_min(float *a, int n)
{
	// each block should cover blockDim.x * 2 elements
	int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
	int s;
	for(s=1; s<blockDim.x; s*=2)
	{
		if (thread_id % 2*s == 0 && thread_id * s + s < n)
		{
			if(a[thread_id * s + s] < a[thread_id * s])
			{
				a[thread_id * s] = a[thread_id * s + s];
			}
		}
		__syncthreads();
	}
}

int main( int argc, char* argv[] )
{
	hipEvent_t start,stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	if(argc < 2)
	{
   		printf("need exactly 1 argument\n");
    		return 0;
  	}
  	int vector_size = atoi(argv[1]);
	
	float* host_a = (float*)malloc(sizeof(float)*vector_size);
	
	float* device_a;
	hipMalloc(&device_a, sizeof(float)*vector_size);
	
	int i;
	for(i=0; i<vector_size; i++)
	{
		host_a[i] = rand() % vector_size;
	}
	
	hipEventRecord(start,0);

	hipMemcpy( device_a, host_a, sizeof(float)*vector_size, hipMemcpyHostToDevice);

	int block_size = 1024;
	int num_blocks = vector_size/block_size;
	if(vector_size%block_size)
		num_blocks++;

	vector_min<<<num_blocks, block_size>>>(device_a, vector_size);
	
	hipMemcpy( host_a, device_a, sizeof(float)*vector_size, hipMemcpyDeviceToHost);

	for(i=0; i<vector_size; i+= block_size)
	{
		if(host_a[i] < host_a[0])
		{
			host_a[0] = host_a[i];
		}
	}

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);

	printf("%f\n", elapsedTime);

	hipFree(device_a);
	
	free(host_a);
	
	return 0;
}