
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void vector_add(float *a, float *b, float *c, int n)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(tid < n)
		c[tid] = a[tid] + b[tid];
}

int main( int argc, char* argv[] )
{
	hipEvent_t start,stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	if(argc < 2)
	{
   		printf("need exactly 1 argument\n");
    	return 0;
  	}
  	int vector_size = atoi(argv[1]);
	
	float* host_a = (float*)malloc(sizeof(float) * vector_size);
	float* host_b = (float*)malloc(sizeof(float) * vector_size);
	float* host_c = (float*)malloc(sizeof(float) * vector_size);
	
	float* device_a;
	float* device_b;
	float* device_c;
	hipMalloc(&device_a, sizeof(float)*vector_size);
	hipMalloc(&device_b, sizeof(float)*vector_size);
	hipMalloc(&device_c, sizeof(float)*vector_size);
	
	int i;
	for(i=0; i<vector_size; i++)
	{
		host_a[i] = 1;
		host_b[i] = 1;
		//host_a[i] = rand() % vector_size;
		//host_b[i] = rand() % vector_size;
	}

	hipEventRecord(start,0);
	
	hipMemcpy( device_a, host_a, sizeof(float)*vector_size, hipMemcpyHostToDevice);
	hipMemcpy( device_b, host_a, sizeof(float)*vector_size, hipMemcpyHostToDevice);
	
	int block_size = 1024;
	int grid_size = vector_size / block_size;
	if(vector_size % block_size)
	{
		grid_size = grid_size + 1;
	}

	vector_add<<<grid_size, block_size>>>(device_a, device_b, device_c, vector_size);
	
	hipMemcpy( host_c, device_c, sizeof(float)*vector_size, hipMemcpyDeviceToHost);

	float sum = 0;
	for(i=0; i<vector_size; i++)
	{
		sum += host_c[i];
	}

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);

	printf("%f\n", elapsedTime);

	hipFree(device_a);
	hipFree(device_b);
	hipFree(device_c);
	
	free(host_a);
	free(host_b);
	free(host_c);
	
	return 0;
}